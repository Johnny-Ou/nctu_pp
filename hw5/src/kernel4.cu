#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int width, int* img, int maxIterations, int pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;
    
    // img[nidex] = mandel(x, y, maxIterations);
    float tempX = x;
    float tempY = y;
    int i;
    for (i = 0; i < maxIterations; ++i)
    {

        if (tempX * tempX + tempY * tempY > 4.f)
        break;

        float new_tempX = tempX * tempX - tempY * tempY;
        float new_tempY = 2.f * tempX * tempY;
        tempX = x + new_tempX;
        tempY = y + new_tempY;
    }
    
    int* row = (int *)((char*)img + thisY * pitch);
    row[thisX] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *h_img, *d_img;
    size_t pitch;

    // cudaHostAlloc((void **)&h_img, resX * resY * sizeof(int), cudaHostAllocDefault);
    hipMallocPitch((void **)&d_img, &pitch, resX * sizeof(int), resY);

    dim3 threadsPerBlock(8, 8);
    dim3 numBlock(resX/threadsPerBlock.x, resY/threadsPerBlock.y);

    mandelKernel<<<numBlock, threadsPerBlock>>>(lowerX, lowerY, stepX, stepY, resX, d_img, maxIterations, pitch);

    hipMemcpy2D(img, resX * sizeof(int), d_img, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    // memcpy(img, h_img, resX * resY * sizeof(int));

    // cudaFreeHost(h_img);
    hipFree(d_img);
}