#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void convolution(float *inputImage, float *outputImage, float *filter,
                            int imageHeight, int imageWidth, int filterWidth)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int halffilterSize = filterWidth / 2;
    int k, l;
    float sum = 0.0f;

    for (k = -halffilterSize; k <= halffilterSize; k++)
    {
        for (l = -halffilterSize; l <= halffilterSize; l++)
        {
            if(filter[(k + halffilterSize) * filterWidth + l + halffilterSize] != 0)
            {
                if (row + k >= 0 && row + k < imageHeight &&
                    col + l >= 0 && col + l < imageWidth)
                {
                    sum += inputImage[(row + k) * imageWidth + col + l] *
                            filter[(k + halffilterSize) * filterWidth +
                                    l + halffilterSize];
                }
            }
        }
    }
    outputImage[row * imageWidth + col] = sum;
}
void hostFE(int filterWidth, float *filter, int imageHeight, int imageWidth, float *inputImage, float *outputImage)
{
    float *d_inputImage;
    float *d_outputImage;
    float *d_filter;
    int imageSize = imageHeight * imageWidth;
    int filterSize = filterWidth * filterWidth;

    hipMalloc((void **)&d_inputImage, imageSize * sizeof(float));
    hipMalloc((void **)&d_outputImage, imageSize * sizeof(float));
    hipMalloc((void **)&d_filter,  filterSize * sizeof(float));

    hipMemcpy(d_inputImage, inputImage, imageSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter, filterSize * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlock(imageWidth/threadsPerBlock.x, imageHeight/threadsPerBlock.y);

    convolution<<<numBlock, threadsPerBlock>>>(d_inputImage, d_outputImage, d_filter, imageHeight, imageWidth, filterWidth);
    hipMemcpy(outputImage, d_outputImage, imageSize * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_inputImage);
    hipFree(d_outputImage);
    hipFree(d_filter);
}